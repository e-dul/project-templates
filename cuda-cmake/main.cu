#include <iostream>
#include <vector>
#include <numeric>

#include <hip/hip_runtime.h>

__global__ void cudaKernel(int* array, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        array[idx] = array[idx] * 2;
    }
}


int main(int argc, char* argv[]) {
    // if (argc != 2) {
    //     std::cerr << "Usage: " << argv[0] << " <path_to_dataset>" << std::endl;
    //     return 1;
    // }

    // const std::string imagePath = argv[1];

    // Read dataset
    std::vector<int> host_data(10);
    std::iota(host_data.begin(), host_data.end(), 0);

    std::cout << "Double input values: " << std::endl;
    for (auto &val : host_data) {
        std::cout << std::to_string(val) << " ";
    }
    std::cout << std::endl;

    int* device_data = nullptr;
    auto size = host_data.size();

    hipMalloc(&device_data, size * sizeof(int));
    hipMemcpy(device_data, host_data.data(), size * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    cudaKernel<<<numBlocks, blockSize>>>(device_data, size);
    hipDeviceSynchronize();

    hipMemcpy(host_data.data(), device_data, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_data);

    std::cout << "Result: " << std::endl;
    for (auto &val : host_data) {
        std::cout << std::to_string(val) << " ";
    }
    std::cout << std::endl;

    return 0;
}

