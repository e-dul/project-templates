#include "hip/hip_runtime.h"
#include "kernel.h"

#include <hip/hip_runtime.h>

__global__ void cudaKernel(int* array, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        array[idx] = array[idx] * 2;
    }
}

void launchCudaKernel(std::vector<int> & data) {
    int* device_data = nullptr;
    auto size = data.size();

    hipMalloc(&device_data, size * sizeof(int));
    hipMemcpy(device_data, data.data(), size * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    cudaKernel<<<numBlocks, blockSize>>>(device_data, size);
    hipDeviceSynchronize();

    hipMemcpy(data.data(), device_data, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_data);
}
